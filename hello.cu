#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc hello.cu -o hello -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
   printf("Hello world\n");
}

// CHANGE: Initial main
// int main()
// {
//   hello<<<1,10>>>();
//   hipDeviceSynchronize();
//   return 0;
// }

// CHANGE: Running hello with 2 blocks of 1024 threads
// int main()
// {
//   hello<<<2,1024>>>();
//   hipDeviceSynchronize();
//   return 0;
// }


//CHANGE: Running hello with 1 block of 2048 threads
int main()
{
  hello<<<1,2048>>>();
  hipDeviceSynchronize();
  return 0;
}
